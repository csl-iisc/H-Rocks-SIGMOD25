#include "debugger.h"
#include <iostream>
#include "gmemtable.h"
#include "gpu_updates.cuh"

bool string_compare(const char* a, const char* b, size_t length); 


struct string_comparator {
    const char* data;
    int keyLength;

    string_comparator(const char* _data, int _keyLength) : data(_data), keyLength(_keyLength) {}

    __device__ bool operator()(const int& a, const int& b) const {
        const char* keyA = data + a * keyLength;
        const char* keyB = data + b * keyLength;

        for (int i = 0; i < keyLength; ++i) {
            if (keyA[i] != keyB[i])
                return keyA[i] < keyB[i];
        }
        return false; // Return false if they are equal
    }
};


void GpuUpdates::sortSmallerUpdates(uint64_t* gIndices) {

    int keyLength = (*activeTable)->keyLength;
    uint64_t numUpdates = (*activeTable)->numKeys; 
    debug.print("numUpdates: " + std::to_string(numUpdates) + " KeyLength: " + std::to_string(keyLength));
    thrust::device_vector<uint64_t> indices(numUpdates);

    char* keys = batch->keys; 
    thrust::device_vector<char> dKeys(numUpdates * keyLength);
    hipMemcpyAsync(thrust::raw_pointer_cast(dKeys.data()), keys, numUpdates * keyLength * sizeof(char), hipMemcpyDeviceToDevice);
    // thrust::device_vector<int> indices(numUpdates);
    thrust::sequence(indices.begin(), indices.end());

    hipError_t err = hipPeekAtLastError();
    debug.print("Memory operations and setup error: " + std::to_string(err));

    // Sort using custom comparator
    thrust::sort(
        thrust::device,
        indices.begin(),
        indices.end(),
        string_comparator(thrust::raw_pointer_cast(dKeys.data()), keyLength)
    );

    err = hipPeekAtLastError();
    debug.print("Thrust sort Error: " + std::to_string(err));

    hipMemcpy(gIndices, thrust::raw_pointer_cast(indices.data()), numUpdates * sizeof(uint64_t), hipMemcpyDeviceToDevice);
    
    err = hipPeekAtLastError();
    debug.print("Memcpy sort Error: " + std::to_string(err));


#if 0
    char* sorted_keys = new char[numUpdates * keyLength];
    // copy the indices
    uint64_t* hIndices = new uint64_t[numUpdates];
    hipMemcpy(hIndices, gIndices, numUpdates * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(sorted_keys, thrust::raw_pointer_cast(dKeys.data()), numUpdates * keyLength * sizeof(char), hipMemcpyDeviceToHost);
    std::cout << "Sorted Keys:\n";
    for (int i = 0; i < numUpdates; i++) {
        std::cout << "i: " << i << " sorted index: " << hIndices[i] << " " << &sorted_keys[hIndices[i] * keyLength] << "\n";
    }
    delete[] sorted_keys;
#endif

}