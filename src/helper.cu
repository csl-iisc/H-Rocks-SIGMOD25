#include "helper.cuh"
#include "debugger.h"
#include <string>

Debugger debug(DEBUG); 

void allocateMemoryManaged(void** ptr, size_t size) {
    hipMallocManaged(ptr, size);
    debug.print("Allocated memory for pointer: " + std::string(typeid(*ptr).name()) + " with size: " + std::to_string(size) + " bytes");
    hipError_t err = hipPeekAtLastError();
    debug.print("Error: " + std::to_string(err)); 
}

void allocateMemory(void** ptr, size_t size) {
    hipMalloc(ptr, size);    
    debug.print("Allocated memory for pointer: " + std::string(typeid(*ptr).name()) + " with size: " + std::to_string(size) + " bytes");
    hipError_t err = hipPeekAtLastError(); 
    debug.print("Error: " + std::to_string(err)); 
}

void freeMemory(void* ptr) {
    if (!ptr) {
        debug.print("Pointer is null, nothing to free.");
        return;
    }
    hipFree(ptr);
    debug.print("Freed memory for pointer: " + std::string(typeid(ptr).name()));
    hipError_t err = hipPeekAtLastError(); 
    debug.print("Error: " + std::to_string(err)); 
}

void allocateMemoryHost(void** ptr, size_t size) {
    hipHostMalloc(ptr, size);
    debug.print("Allocated host memory for pointer: " + std::string(typeid(*ptr).name()) + " with size: " + std::to_string(size) + " bytes");
    hipError_t err = hipPeekAtLastError(); 
    debug.print("Error: " + std::to_string(err)); 
}

void copyMemory(void* dst, void* src, size_t size, hipMemcpyKind kind) {
    hipMemcpy(dst, src, size, kind);
    debug.print("Copied memory from " + std::string(typeid(src).name()) + " to " + std::string(typeid(dst).name()) + " with size: " + std::to_string(size) + " bytes");
    hipError_t err = hipPeekAtLastError(); 
    debug.print("Error: " + std::to_string(err)); 
}

void copyMemoryAsync(void* dst, void* src, size_t size, hipMemcpyKind kind) {
    hipMemcpyAsync(dst, src, size, kind);
    debug.print("Copied memory async from " + std::string(typeid(src).name()) + " to " + std::string(typeid(dst).name()) + " with size: " + std::to_string(size) + " bytes");
    hipError_t err = hipPeekAtLastError(); 
    debug.print("Error: " + std::to_string(err)); 
}

void synchronizeDevice() {
    hipDeviceSynchronize();
    debug.print("Synchronized device");
    hipError_t err = hipPeekAtLastError(); 
    debug.print("Error: " + std::to_string(err)); 
}

void setDevice(int device) {
    hipSetDevice(device);
    hipError_t err = hipPeekAtLastError(); 
    debug.print("Error: " + std::to_string(err)); 
}

void getDeviceCount(int* count) {
    hipGetDeviceCount(count);
    hipError_t err = hipPeekAtLastError(); 
    debug.print("Error: " + std::to_string(err)); 
}

void registerHost(void* ptr, size_t size) {
    hipHostRegister(ptr, size, 0);
    debug.print("Registered host memory for pointer: " + std::string(typeid(ptr).name()) + " with size: " + std::to_string(size) + " bytes");
    hipError_t err = hipPeekAtLastError(); 
    debug.print("Error: " + std::to_string(err)); 
}